#include "hip/hip_runtime.h"
#include <stdio.h>
#include "utils.h"

__global__ void compute_acceleration(vector_t *d_acc_arr, vector_t *d_v_arr, 
                        vector_t *d_pos_arr, float *d_m_arr, int n, float grav) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= n) return;
    d_acc_arr[i].x = 0;
    d_acc_arr[i].y = 0;
    for (int j = 0; j < n; ++j) {
        if (j == i) continue;
        // compute force
        float dis2 = powf(d_pos_arr[i].x - d_pos_arr[j].x, 2) + powf(d_pos_arr[i].y - d_pos_arr[j].y, 2);
        float dis = sqrtf(dis2);
        float fx = 0, fy = 0;
        if (dis != 0.0f) {
            float f = grav * d_m_arr[i] * d_m_arr[j] / dis2;
            fx = f * (d_pos_arr[j].x - d_pos_arr[i].x) / dis;
            fy = f * (d_pos_arr[j].y - d_pos_arr[i].y) / dis;
        }
        // update acceleration
        d_acc_arr[i].x = fx / d_m_arr[i];
        d_acc_arr[i].y = fy / d_m_arr[i];
    }
}

__global__ void update_velocity_position(vector_t *d_acc_arr, vector_t *d_v_arr, 
                    vector_t *d_pos_arr, float *d_m_arr, int n, float dt) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= n) return;

    d_v_arr[i].x += d_acc_arr[i].x * dt;
    d_v_arr[i].y += d_acc_arr[i].y * dt;
    d_pos_arr[i].x += d_v_arr[i].x * dt;
    d_pos_arr[i].y += d_v_arr[i].y * dt;
}


extern "C" void __nbody_cuda_single_naive(int n, int m, float dt, particle_t parts[], float grav, FILE* fp, bool full_output) {
    int m_size, m_rank;
    MPI_Comm_size(MPI_COMM_WORLD, &m_size);
    MPI_Comm_rank(MPI_COMM_WORLD, &m_rank);
    if (m_rank != ROOT_NODE) return;

    // vector_t *acc_arr = (vector_t *) malloc(sizeof(vector_t) * n);
    vector_t *v_arr = (vector_t *) malloc(sizeof(vector_t) * n);
    vector_t *pos_arr = (vector_t *) malloc(sizeof(vector_t) * n);
    float *m_arr = (float *) malloc(sizeof(float) * n);

    for (int i = 0; i < n; ++i) {
        v_arr[i] = parts[i].v;
        pos_arr[i] = parts[i].pos;
        m_arr[i] = parts[i].mass;
    }

    vector_t *d_acc_arr = 0;
    vector_t *d_v_arr = 0;
    vector_t *d_pos_arr = 0;
    float *d_m_arr = 0;
    hipMalloc((void**)&d_acc_arr, sizeof(vector_t) * n);
    hipMalloc((void**)&d_v_arr, sizeof(vector_t) * n);
    hipMalloc((void**)&d_pos_arr, sizeof(vector_t) * n);
    hipMalloc((void**)&d_m_arr, sizeof(float) * n);
    
    hipMemcpy(d_v_arr, v_arr, sizeof(vector_t) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_pos_arr, pos_arr, sizeof(vector_t) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_m_arr, m_arr, sizeof(float) * n, hipMemcpyHostToDevice);

    const int threadsPerBlock = 256;
    int threadsPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    

    for (int m_i = 0; m_i < m; ++m_i) {

        compute_acceleration<<<threadsPerGrid, threadsPerBlock>>>
                    (d_acc_arr, d_v_arr, d_pos_arr, d_m_arr, n, grav);

        update_velocity_position<<<threadsPerGrid, threadsPerBlock>>>
                    (d_acc_arr, d_v_arr, d_pos_arr, d_m_arr, n, dt);

        // last step or full output mode
        if ((m_rank == ROOT_NODE) && (m_i == (m - 1) || full_output)) {
            hipMemcpy(pos_arr, d_pos_arr, sizeof(vector_t) * n, hipMemcpyDeviceToHost);
            output_particle_pos_v(n, pos_arr, fp);
        }
    }


    hipFree(d_acc_arr);
    hipFree(d_v_arr);
    hipFree(d_pos_arr);
    hipFree(d_m_arr);

    free(v_arr);
    free(pos_arr);
    free(m_arr);
}
