#include "hip/hip_runtime.h"
#include <stdio.h>
#include "utils.hpp"
#include "nbody_cuda.hpp"

__global__ void compute_acceleration_local(vector_t *d_l_acc_arr, vector_t *d_pos_arr, 
                    float *d_m_arr, int n, int local_n, int offset, float grav) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= local_n) return;
    d_l_acc_arr[i].x = 0;
    d_l_acc_arr[i].y = 0;
    int gi = i + offset;
    for (int j = 0; j < n; ++j) {
        if (j == gi) continue;
        // compute force
        float dis2 = powf(d_pos_arr[gi].x - d_pos_arr[j].x, 2) + powf(d_pos_arr[gi].y - d_pos_arr[j].y, 2);
        float dis = sqrtf(dis2);
        float fx = 0, fy = 0;
        if (dis != 0.0f) {
            float f = grav * d_m_arr[gi] * d_m_arr[j] / dis2;
            fx = f * (d_pos_arr[j].x - d_pos_arr[gi].x) / dis;
            fy = f * (d_pos_arr[j].y - d_pos_arr[gi].y) / dis;
        }
        // update acceleration
        d_l_acc_arr[i].x = fx / d_m_arr[gi];
        d_l_acc_arr[i].y = fy / d_m_arr[gi];
    }
}

__global__ void update_velocity_position_local(vector_t *d_l_acc_arr, vector_t *d_l_v_arr, 
                    vector_t *d_pos_arr, float *d_m_arr, int n, int local_n, int offset, float dt) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= local_n) return;
    int gi = i + offset;

    d_l_v_arr[i].x += d_l_acc_arr[i].x * dt;
    d_l_v_arr[i].y += d_l_acc_arr[i].y * dt;
    d_pos_arr[gi].x += d_l_v_arr[i].x * dt;
    d_pos_arr[gi].y += d_l_v_arr[i].y * dt;
}

// The code of following two methods are very similar.
// We duplicate the code instead of reusing them 
// in order to specialize and optimize in the future.
// ===============================================================================
// nbody_cuda_mpi_naive
// ===============================================================================

void nbody_cuda_mpi_naive(int n, int m, float dt, particle_t parts[], float grav, FILE* fp, bool full_output) {
    int m_size, m_rank;
    MPI_Comm_size(MPI_COMM_WORLD, &m_size);
    MPI_Comm_rank(MPI_COMM_WORLD, &m_rank);

    if (m_rank == ROOT_NODE) {
        printf("MPI_size: %d\n", m_size);
    }

    // distribute work
    int n_per_node = n / m_size, extra_n = n % m_size;
    int interval_of_nodes[m_size + 1];
    interval_of_nodes[0] = 0;
    for (int i = 0; i < m_size; ++i) {
        // first extra_n nodes will have an extra items`
        interval_of_nodes[i+1] = interval_of_nodes[i] + n_per_node + (i < extra_n);
    }

    // [local_l, local_r)
    int local_l = interval_of_nodes[m_rank], local_r = interval_of_nodes[m_rank + 1];
    int local_n = local_r - local_l;
    vector_t *local_acc = (vector_t *) malloc(sizeof(vector_t) * local_n);
    vector_t *local_v = (vector_t *) malloc(sizeof(vector_t) * local_n);


    // // vector_t *acc_arr = (vector_t *) malloc(sizeof(vector_t) * n);
    vector_t *local_v_arr = (vector_t *) malloc(sizeof(vector_t) * local_n);
    vector_t *pos_arr = (vector_t *) malloc(sizeof(vector_t) * n);
    float *m_arr = (float *) malloc(sizeof(float) * n);

    // copy data
    for (int i = 0; i < n; ++i) {
        pos_arr[i] = parts[i].pos;
        m_arr[i] = parts[i].mass;
    }
    for (int i = 0; i < local_n; ++i) local_v_arr[i] = parts[i + local_l].v;

    vector_t *d_local_acc_arr = 0;
    vector_t *d_local_v_arr = 0;
    vector_t *d_pos_arr = 0;
    float *d_m_arr = 0;
    hipMalloc((void**)&d_local_acc_arr, sizeof(vector_t) * local_n);
    hipMalloc((void**)&d_local_v_arr, sizeof(vector_t) * local_n);
    hipMalloc((void**)&d_pos_arr, sizeof(vector_t) * n);
    hipMalloc((void**)&d_m_arr, sizeof(float) * n);
    
    hipMemcpy(d_local_v_arr, local_v_arr, sizeof(vector_t) * local_n, hipMemcpyHostToDevice);
    hipMemcpy(d_pos_arr, pos_arr, sizeof(vector_t) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_m_arr, m_arr, sizeof(float) * n, hipMemcpyHostToDevice);

    const int threadsPerBlock = 256;
    int threadsPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    

    for (int m_i = 0; m_i < m; ++m_i) {

        compute_acceleration_local<<<threadsPerGrid, threadsPerBlock>>>
                (d_local_acc_arr, d_pos_arr, d_m_arr, n, local_n, local_l, grav);

        update_velocity_position_local<<<threadsPerGrid, threadsPerBlock>>>
                (d_local_acc_arr, d_local_v_arr, d_pos_arr, d_m_arr, n, local_n, local_l, dt);

        // only sync the changed part
        vector_t *local_pos_arr = pos_arr + local_l;
        vector_t *d_local_pos_arr = d_pos_arr + local_l;
        hipMemcpy(local_pos_arr, d_local_pos_arr, sizeof(vector_t) * local_n, hipMemcpyDeviceToHost);
        // broadcast pos_arr
        for (int i = 0; i < m_size; ++i) {
            int l = interval_of_nodes[i];
            int r = interval_of_nodes[i + 1];
            MPI_Bcast(pos_arr + l, r - l, mpi_vector_t, i, MPI_COMM_WORLD);
        }
        hipMemcpy(d_pos_arr, pos_arr, sizeof(vector_t) * n, hipMemcpyHostToDevice);

        // last step or full output mode
        if ((m_rank == ROOT_NODE) && (m_i == (m - 1) || full_output)) {
            output_particle_pos_v(n, pos_arr, fp);
        }
    }


    hipFree(d_local_acc_arr);
    hipFree(d_local_v_arr);
    hipFree(d_pos_arr);
    hipFree(d_m_arr);

    free(local_v_arr);
    free(pos_arr);
    free(m_arr);
}


// ===============================================================================
// nbody_cuda_single_naive
// ===============================================================================

void nbody_cuda_single_naive(int n, int m, float dt, particle_t parts[], float grav, FILE* fp, bool full_output) {
    int m_size, m_rank;
    MPI_Comm_size(MPI_COMM_WORLD, &m_size);
    MPI_Comm_rank(MPI_COMM_WORLD, &m_rank);
    if (m_rank != ROOT_NODE) return;

    // vector_t *acc_arr = (vector_t *) malloc(sizeof(vector_t) * n);
    vector_t *v_arr = (vector_t *) malloc(sizeof(vector_t) * n);
    vector_t *pos_arr = (vector_t *) malloc(sizeof(vector_t) * n);
    float *m_arr = (float *) malloc(sizeof(float) * n);

    for (int i = 0; i < n; ++i) {
        v_arr[i] = parts[i].v;
        pos_arr[i] = parts[i].pos;
        m_arr[i] = parts[i].mass;
    }

    vector_t *d_acc_arr = 0;
    vector_t *d_v_arr = 0;
    vector_t *d_pos_arr = 0;
    float *d_m_arr = 0;
    hipMalloc((void**)&d_acc_arr, sizeof(vector_t) * n);
    hipMalloc((void**)&d_v_arr, sizeof(vector_t) * n);
    hipMalloc((void**)&d_pos_arr, sizeof(vector_t) * n);
    hipMalloc((void**)&d_m_arr, sizeof(float) * n);
    
    hipMemcpy(d_v_arr, v_arr, sizeof(vector_t) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_pos_arr, pos_arr, sizeof(vector_t) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_m_arr, m_arr, sizeof(float) * n, hipMemcpyHostToDevice);

    const int threadsPerBlock = 256;
    int threadsPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    

    for (int m_i = 0; m_i < m; ++m_i) {

        compute_acceleration_local<<<threadsPerGrid, threadsPerBlock>>>
                    (d_acc_arr, d_pos_arr, d_m_arr, n, n, 0, grav);

        update_velocity_position_local<<<threadsPerGrid, threadsPerBlock>>>
                    (d_acc_arr, d_v_arr, d_pos_arr, d_m_arr, n, n, 0, dt);

        // last step or full output mode
        if ((m_rank == ROOT_NODE) && (m_i == (m - 1) || full_output)) {
            hipMemcpy(pos_arr, d_pos_arr, sizeof(vector_t) * n, hipMemcpyDeviceToHost);
            output_particle_pos_v(n, pos_arr, fp);
        }
    }


    hipFree(d_acc_arr);
    hipFree(d_v_arr);
    hipFree(d_pos_arr);
    hipFree(d_m_arr);

    free(v_arr);
    free(pos_arr);
    free(m_arr);
}
