#include "hip/hip_runtime.h"
#include <stdio.h>
#include "utils.h"

__global__ void compute_acceleration(vector_t *d_acc_arr, vector_t *d_v_arr, 
                        vector_t *d_pos_arr, float *d_m_arr, int n, float dt, float grav) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= n) return;
    d_acc_arr[i].x = 0;
    d_acc_arr[i].y = 0;
    for (int j = 0; j < n; ++j) {
        if (j == i) continue;
        // compute force
        float dis2 = powf(d_pos_arr[i].x - d_pos_arr[j].x, 2) + powf(d_pos_arr[i].y - d_pos_arr[j].y, 2);
        float dis = sqrtf(dis2);
        float fx = 0, fy = 0;
        if (dis != 0.0f) {
            float f = grav * d_m_arr[i] * d_m_arr[j] / dis2;
            fx = f * (d_pos_arr[j].x - d_pos_arr[i].x) / dis;
            fy = f * (d_pos_arr[j].y - d_pos_arr[i].y) / dis;
        }
        // update acceleration
        d_acc_arr[i].x = fx / d_m_arr[i];
        d_acc_arr[i].y = fy / d_m_arr[i];
    }
}


extern "C" void __nbody_cuda_single_naive(int n, int m, float dt, particle_t parts[], float grav, FILE* fp, bool full_output) {
    int m_size, m_rank;
    MPI_Comm_size(MPI_COMM_WORLD, &m_size);
    MPI_Comm_rank(MPI_COMM_WORLD, &m_rank);
    if (m_rank != ROOT_NODE) return;

    // vector_t *acc_arr = (vector_t *) malloc(sizeof(vector_t) * n);
    vector_t *v_arr = (vector_t *) malloc(sizeof(vector_t) * n);
    vector_t *pos_arr = (vector_t *) malloc(sizeof(vector_t) * n);
    float *m_arr = (float *) malloc(sizeof(float) * n);

    for (int i = 0; i < n; ++i) {
        v_arr[i] = parts[i].v;
        pos_arr[i] = parts[i].pos;
        m_arr[i] = parts[i].m;
    }

    vector_t *d_acc_arr = 0;
    vector_t *d_v_arr = 0;
    vector_t *d_pos_arr = 0;
    float *d_m_arr = 0;
    hipMalloc((void**)&d_acc_arr, sizeof(vector_t) * n);
    hipMalloc((void**)&d_v_arr, sizeof(vector_t) * n);
    hipMalloc((void**)&d_pos_arr, sizeof(vector_t) * n);
    hipMalloc((void**)&d_m_arr, sizeof(float) * n);
    
    hipMemcpy(d_v_arr, v_arr, sizeof(vector_t) * n, cudaMemcopyHostToDevice);
    hipMemcpy(d_pos_arr, pos_arr, sizeof(vector_t) * n, cudaMemcopyHostToDevice);
    hipMemcpy(d_m_arr, m_arr, sizeof(float) * n, cudaMemcopyHostToDevice);

    for (int m_i = 0; m_i < m; ++m_i) {
        // // reset acceleration
        // #pragma omp parallel for 
        // for (int i = 0; i < local_n; ++i) {
        //     local_acc[i].x = 0;
        //     local_acc[i].y = 0;
        // }

        // // compute acceleration
        // #pragma omp parallel for 
        // for (int i = 0; i < local_n; ++i) {
        //     int global_i = i + local_l;
        //     for (int j = 0; j < n; ++j) {
        //         if (j == global_i) continue;
        //         vector_t f = force_between_particle(pos_arr[global_i], pos_arr[j], parts[global_i].mass, parts[j].mass, grav);
        //         local_acc[i].x += f.x / parts[global_i].mass;
        //         local_acc[i].y += f.y / parts[global_i].mass;
        //     }
        // }

        

        // // update velocity & position
        // #pragma omp parallel for 
        // for (int i = 0; i < local_n; ++i) {
        //     int global_i = i + local_l;
        //     // printf("%d\n", global_i);
        //     local_v[i].x += local_acc[i].x * dt;
        //     local_v[i].y += local_acc[i].y * dt;
        //     pos_arr[global_i].x += local_v[i].x * dt;
        //     pos_arr[global_i].y += local_v[i].y * dt;
        // }


        // last step or full output mode
        if ((m_rank == ROOT_NODE) && (m_i == (m - 1) || full_output)) {
            hipMemcpy(pos_arr, d_pos_arr, sizeof(vector_t) * n, hipMemcpyDeviceToHost);
            output_particle_pos_v(n, pos_arr, fp);
        }

    }


    hipFree(d_acc_arr);
    hipFree(d_v_arr);
    hipFree(d_pos_arr);
    hipFree(d_m_arr);

    free(v_arr);
    free(pos_arr);
    free(m_arr);
}
